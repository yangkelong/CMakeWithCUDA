#include "hip/hip_runtime.h"
#include "add.cuh"

 __global__ void addCUDA(int *a, int *b, int *c, const int n) {
     int i = threadIdx.x + blockIdx.x * blockDim.x;
     if (i < n) {
         c[i] = a[i] + b[i];
     }
 }
 
 void addgpu(int *a, int *b, int *c, int n) {
     int *d_a, *d_b, *d_c;
 
     // allocate device memory
     hipMalloc(&d_a, n*sizeof(int));
     hipMalloc(&d_b, n*sizeof(int));
     hipMalloc(&d_c, n*sizeof(int));
 
     // copy data from host to device
     hipMemcpy(d_a, a, n*sizeof(int), hipMemcpyHostToDevice);
     hipMemcpy(d_b, b, n*sizeof(int), hipMemcpyHostToDevice);
 
     // launch addGpu kernel
     int blockSize = 256;
     int gridSize = (n + blockSize - 1) / blockSize;
     addCUDA<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
 
     // copy result from device to host
     hipMemcpy(c, d_c, n*sizeof(int), hipMemcpyDeviceToHost);
 
     // free device memory
     hipFree(d_a);
     hipFree(d_b);
     hipFree(d_c);
 }