#ifndef _KERNEL_SUPPORT_
#define _KERNEL_SUPPORT_

#include <iostream>


__global__ void TestDevice(int *deviceArray)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    deviceArray[idx] = deviceArray[idx]*deviceArray[idx];
}

int main( int argc, char** argv) 
{
    int* hostArray;
    int* deviceArray;
    const int arrayLength = 16;
    const unsigned int memSize = sizeof(int) * arrayLength;

    hostArray = (int*)malloc(memSize);
    hipMalloc((void**) &deviceArray, memSize);

    std::cout << "Before device\n";
    for(int i=0;i<arrayLength;i++)
    {
        hostArray[i] = i+1;
        std::cout << hostArray[i] << "\n";
    }
    std::cout << "\n";

    hipMemcpy(deviceArray, hostArray, memSize, hipMemcpyHostToDevice);
    TestDevice <<< 4, 4 >>> (deviceArray);
    hipMemcpy(hostArray, deviceArray, memSize, hipMemcpyDeviceToHost);

    std::cout << "After device\n";
    for(int i=0;i<arrayLength;i++)
    {
        std::cout << hostArray[i] << "\n";
    }

    hipFree(deviceArray);
    free(hostArray);

    std::cout << "Done\n";
}

#endif