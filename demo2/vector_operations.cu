#include "hip/hip_runtime.h"
// vector_operations.cpp
#include "vector_operations.h"
#include <hip/hip_runtime.h>
#include <>


extern "C" __global__ void squareKernel(float* input, float* output, int numElements) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numElements) {
        output[i] = input[i] * input[i];
    }
}

extern "C" __host__ void squareElements(float* input, float* output, int numElements) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    float* d_input = nullptr;
    float* d_output = nullptr;

    hipMalloc(&d_input, numElements * sizeof(float));
    hipMalloc(&d_output, numElements * sizeof(float));
    hipMemcpy(d_input, input, numElements * sizeof(float), hipMemcpyHostToDevice);

    squareKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, numElements);

    hipMemcpy(output, d_output, numElements * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}